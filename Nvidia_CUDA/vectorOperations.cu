//#include <__clang_cuda_builtin_vars.h>
#include <iostream>
#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void vectorAdd(int *a,int *b,int *c )
{
	int i = threadIdx.x; 
	c[i] = a[i]+b[i];
	return;
}
int main()
{
	int a[3] = {5,2,3};
	int b[3] = {4,5,6};
	int c[sizeof(a)/sizeof(int)] = {0,0,0};
	//for (int i=0 ; i<sizeof(c); i++) {
	//	c[i] = a[i] + b[i];
	//}

	int *cudaA=0;
	int *cudaB=0;
	int *cudaC=0;

	hipMalloc(&cudaA,sizeof(a));
	hipMalloc(&cudaB,sizeof(b));
	hipMalloc(&cudaC,sizeof(c));
	
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
	//cudaMemcpy(cudaC, c, sizeof(c), cudaMemcpyHostToDevice);

	vectorAdd <<< 1,sizeof(a)/sizeof(int) >>> (cudaA,cudaB,cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
	for (int i : c) {
		std::cout << i << " " ;
	}
	
}
